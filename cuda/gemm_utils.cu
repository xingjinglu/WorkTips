#include "gemm_utils.h"

void  checkKernelsErrors(const char *prefix, const char *postfix) 
{       
  hipDeviceSynchronize();
  if(hipPeekAtLastError() != hipSuccess){                                
    printf("\n%s Line %d: %s %s\n", prefix, __LINE__, 
        hipGetErrorString(hipGetLastError()),   
        postfix);                                                          
    hipDeviceReset();                                                     
    exit(1);                                                               
  }                                                                        

  return;
} 
