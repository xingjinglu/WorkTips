#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void MyKernel(int *array, int arrayCount) 
{ 
  int idx = threadIdx.x + blockIdx.x * blockDim.x; 
  if (idx < arrayCount) 
  { 
    array[idx] *= array[idx]; 
  } 
} 

void launchMyKernel(int *array, int arrayCount) 
{ 
  int blockSize;   // The launch configurator returned block size 
  int minGridSize; // The minimum grid size needed to achieve the 
  // maximum occupancy for a full device launch 
  int gridSize;    // The actual grid size needed, based on input size 

  hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, 
      MyKernel, 0, 0); 
  // Round up according to array size 
  gridSize = (arrayCount + blockSize - 1) / blockSize; 

  MyKernel<<< gridSize, blockSize >>>(array, arrayCount); 

  hipDeviceSynchronize(); 

  // calculate theoretical occupancy
  int maxActiveBlocks;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &maxActiveBlocks, 
      MyKernel,
      blockSize, 
      0);

  int device;
  hipDeviceProp_t props;
  hipGetDevice(&device);
  hipGetDeviceProperties(&props, device);

  float occupancy = (maxActiveBlocks * blockSize /
      props.warpSize) / 
    (float)(props.maxThreadsPerMultiProcessor
        / 
        props.warpSize);

  printf("Launched blocks of size %d. Theoretical
      occupancy: %f\n", 
      blockSize, occupancy);
}
